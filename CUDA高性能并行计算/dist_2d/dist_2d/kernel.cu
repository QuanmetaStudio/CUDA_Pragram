#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<GL/glew.h>

#define W 500
#define H 500
#define TX 32		// number of threads per block along x-axis
#define TY 32

#include<stdlib.h>
#include <stdio.h>

__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__global__
void distanceKernel(uchar4 *d_output, int w, int h, int2 pos)
{
	// uchar4���ĸ���Ա������ͼ����ĸ�ͨ�� 
	// Compute column and row indices
	const int c = blockIdx.x*blockDim.x + threadIdx.x;
	const int r = blockIdx.y*blockDim.y + threadIdx.y;
	const int i = r * w + c;

	// check if within image bounds.
	if ((c >= w) || (r >= h))
		return ;
	// Compute the distance (in pixel spacings)
	const int d = sqrtf((c - pos.x) * (c - pos.x) + (r - pos.y)*(r - pos.y));

	// convert distance to intensity value on interval [0, 255]
	const unsigned char intensity = clip(255 - d);		// ԽԶԽ intensity ԽС

	d_output[i].x = intensity;		// compute red
	d_output[i].y = intensity;		// compute green
	d_output[i].z = 0;		// compute blue
	d_output[i].w = 255;			// Fully opaque
}

int main(void)
{
	uchar4 *out = (uchar4*)calloc(W*H, sizeof(uchar4));		// host memory distributed
	uchar4 *d_out;					// pointer to device memory
	hipMalloc(&d_out, W*H * sizeof(uchar4));  // device memory distributed

	const int2 pos = { 0, 0 };// ���òο���λ��
	const dim3 blockSize(TX, TY); 
	const int bx = (W + TX - 1) / TX;
	const int by = (H + TY - 1) / TY;
	const dim3 gridSize(bx, by);

	distanceKernel << <gridSize, blockSize >> > (d_out, W, H, pos);

	hipMemcpy(out, d_out, W*H * sizeof(uchar4), hipMemcpyDeviceToHost);

	hipFree(d_out);
	free(out);

	return 0;
}
// ʹ��Nsight�鿴�ڴ�