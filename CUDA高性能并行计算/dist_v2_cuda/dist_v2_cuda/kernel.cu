#include "hip/hip_runtime.h"
#include"kernel.h"
//#include<iostream>
//#define TPB 1024
#define TPB 1024

__device__ 
float distance(float x1, float x2) {
	return sqrt((x2 - x1)*(x2 - x1));
}

__global__  // �ں�
void distanceKernel(float* d_out, float* d_in, float ref) {
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = distance(x, ref);
	printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

void distanceArray(float * out, float * in, float ref, float len){
	// declare pointers to device arrays
	float * d_in = NULL;
	float * d_out = NULL;

	// alllocate memory for device arrays
	hipMalloc(&d_in, len * sizeof(float));
	hipMalloc(&d_out, len * sizeof(float));

	// Copy input data from host to device
	hipMemcpy(d_in, in, len * sizeof(float), hipMemcpyHostToDevice);

	//lunch kernel to compute abd store distance values
	distanceKernel <<<(len + TPB - 1) / TPB, TPB >>> (d_out, d_in, ref);
	
	hipMemcpy(out, d_out, len * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
}
