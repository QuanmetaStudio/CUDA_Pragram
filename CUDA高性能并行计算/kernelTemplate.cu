#include "hip/hip_runtime.h"
__global__
void kernel2D(float *d_out, int w, int h, ...)
{
	// Compute column and row indices
	const int c = blockIdx.x*blockDim.x + threadIdx.x;
	const int r = blockIdx.y*blockDim.y + threadIdx.y;
	const int i = r * w + c;

	// check if within image bounds.
	if((c>=w) || (r >= h))
		return error;

	d_out[i] = INSERT_CODE_HERE;  // compute/ store pixel in device array.
}

