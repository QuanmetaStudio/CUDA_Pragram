#include "hip/hip_runtime.h"
#include "kernel.h"
#define TX 32
#define TY 32

// �����￪ʼ������ 
//openGL��һ������װ����

__device__ 
unsigned char clip(int n) {return( n >= 255 ? 255 : (n < 0 ? 0 : n)); }

__global__ 
void distanceKernel(uchar4 *d_out, int w, int h, int2 pos) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x;
	const int r = blockIdx.y * blockDim.y + threadIdx.y;
	if ((c >= w) || (r >= h)) return;
	const int i = r * w + c;
	// important formula
	const int dist = sqrtf((pos.x - c) * (pos.x - c) + (pos.y - r) * (pos.y - r));
	const unsigned char d = clip(255 - dist);
	d_out[i].x = d; // RED
	d_out[i].y = d; // GREEN
	d_out[i].z = 0; // blue
	d_out[i].w = 0;	// opaque
}

void kernelLauncher(uchar4 * d_out, int w, int h, int2 pos)
{
	dim3 blockSize(TX, TY);
	dim3 gridSize((w + TX - 1) / TX, (h + TY - 1) / TY);		// ������ٸ�block
	distanceKernel << <gridSize, blockSize >> > (d_out, w, h, pos);
}
