#include "hip/hip_runtime.h"
__global__
void kernel2D(uchar4 *d_output, int w, int h, ...)
{
	// uchar4的四个成员看作是图像的四个通道 
	// Compute column and row indices
	const int c = blockIdx.x*blockDim.x + threadIdx.x;
	const int r = blockIdx.y*blockDim.y + threadIdx.y;
	const int i = r * w + c;

	// check if within image bounds.
	if((c>=w) || (r >= h))
		return error;

	d_output[i].x = RED_FORMULA;		// compute red
	d_output[i].y = GREEN_FORNULA;		// compute green
	d_output[i].z = BLUE_FORMULA;		// compute blue
	d_output[i].w = 255;			// Fully opaque
}

